
#include <hip/hip_runtime.h>
#include <vector>
#include <chrono>
#include <iostream>
#include <vector>
#include <fstream>

void quadratic_regression(double &a, double &b, double &c,
                          const std::vector<double> &x,
                          const std::vector<double> &y) {
  return;
}

std::vector<std::vector<double>>
generate_random_paths(int n_paths, int n_time_steps, double initial_price,
                      double delta_t, double drift, double volatility) {
                        return {};
                      }


void test(int paths, int steps, double s0, double dt, double strike, double r,
          double drift, double vol, const std::string &save_path)
{
  std::vector<int> stop;
  auto start_time = std::chrono::high_resolution_clock::now();
  auto X = generate_random_paths(paths, steps, s0, dt, drift, vol);
  // // Benchmark the function
  // double price = ls_american_put_option_backward_pass(X, stop, dt, r, strike);
  auto end_time = std::chrono::high_resolution_clock::now();
  std::chrono::duration<double> duration = end_time - start_time;
  // std::cout << "Price: " << price << std::endl;
  std::cout << "Execution time: " << duration.count() << " seconds"
            << std::endl;

  // if (save_path != "")
  // {
  //   std::vector<double> to_save(paths * steps / save_freq);
  //   for (int i = 0; i < steps; i += save_freq)
  //   {
  //     std::copy(X[i].begin(), X[i].end(), to_save.begin() + i / save_freq * paths);
  //   }

  //   std::ofstream outfile(save_path, std::ios::out | std::ios::binary);

  //   if (!outfile)
  //   {
  //     std::cout << "Could not save steps..." << std::endl;
  //     return;
  //   }

  //   outfile.write(reinterpret_cast<const char *>(to_save.data()), to_save.size() * sizeof(double));
  //   outfile.close();
  // }
}

int main(int argc, char *argv[])
{
  // Default values for the parameters
  int paths = 10000;
  int steps = 100;
  double s0 = 100.0;
  double dt = 0.01;
  double strike = 100.0;
  double r = 0.05;
  double drift = 0.05;
  double vol = 0.2;
  std::string save_path = "";

  // Parse command line arguments
  for (int i = 1; i < argc; i++)
  {
    std::string arg = argv[i];
    if (arg == "-paths" && i + 1 < argc)
      paths = std::atoi(argv[++i]);
    else if (arg == "-steps" && i + 1 < argc)
      steps = std::atoi(argv[++i]);
    else if (arg == "-s0" && i + 1 < argc)
      s0 = std::atof(argv[++i]);
    else if (arg == "-dt" && i + 1 < argc)
      dt = std::atof(argv[++i]);
    else if (arg == "-strike" && i + 1 < argc)
      strike = std::atof(argv[++i]);
    else if (arg == "-r" && i + 1 < argc)
      r = std::atof(argv[++i]);
    else if (arg == "-drift" && i + 1 < argc)
      drift = std::atof(argv[++i]);
    else if (arg == "-vol" && i + 1 < argc)
      vol = std::atof(argv[++i]);
    else if (arg == "-save" && i + 1 < argc)
      save_path = argv[++i];
    else
    {
      std::cerr << "Usage: " << argv[0]
                << " [-paths num] [-steps num] [-s0 value] [-dt value] "
                   "[-strike value] [-r rate] [-drift rate] [-vol volatility]"
                << std::endl;
      return 1;
    }
  }

  test(paths, steps, s0, dt, strike, r, drift, vol, save_path);

  return 0;
}
